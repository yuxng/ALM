#include "hip/hip_runtime.h"
/*
  convolution between hog features and hog templates
  author: Yu Xiang
  Date: 04/14/2011
*/

extern "C"
{
#include "convolve.h"
#include "matrix.h"
}
#include <hip/hip_runtime.h>

/* !!! maximum templates size = BLOCK_SIZE * 3 * sbin !!! */
#define BLOCK_SIZE 21

__constant__ float hog_template[2048];
__global__ void convolve2D(CUMATRIX C, CUMATRIX A, CUMATRIX B, int index);

CUMATRIX fconv(CUMATRIX A, CUMATRIX B)
{
  CUMATRIX A_device;
  CUMATRIX B_device;
  CUMATRIX C, C_device;
  hipError_t error;

  A_device = alloc_device_cumatrix(A);
  B_device = alloc_device_cumatrix(B);

  // allocate hog response cumatrix
  C.dims_num = 2;
  C.dims = (int*)malloc(sizeof(int)*2);
  C.dims[0] = A.dims[0];
  C.dims[1] = A.dims[1];
  C.length = C.dims[0]*C.dims[1];
  C.data = (float*)malloc(sizeof(float)*C.length);
  C_device = alloc_device_cumatrix(C);

  error = hipMemset(C_device.data, 0, sizeof(float)*C_device.length);
  if (error != hipSuccess)
  {
    printf("hipMemset C_device returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

  /* setup execution parameters */
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE+2*(B.dims[0]/2));
  dim3 grid((C.dims[1]+BLOCK_SIZE-1) / BLOCK_SIZE, (C.dims[0]+BLOCK_SIZE-1) / BLOCK_SIZE);

  for(int i = 0; i < B.dims[2]; i++)
  {
    // copy to constant memory
    error = hipMemcpyToSymbol(HIP_SYMBOL(hog_template), B.data+i*B.dims[0]*B.dims[1], sizeof(float)*B.dims[0]*B.dims[1]);
    if (error != hipSuccess)
    {
      printf("hipMemcpyToSymbol returned error code %d, line(%d)\n", error, __LINE__);
      exit(EXIT_FAILURE);
    }

    convolve2D<<< grid, threads >>>(C_device, A_device, B_device, i);
    hipDeviceSynchronize();
  }

  /* copy result from device to host */
  error = hipMemcpy(C.data, C_device.data, sizeof(float)*C.length, hipMemcpyDeviceToHost);
  if (error != hipSuccess)
  {
    printf("hipMemcpy C returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

  free_device_cumatrix(&A_device);
  free_device_cumatrix(&B_device);
  free_device_cumatrix(&C_device);
  return C;
}

// implementation of the convolution algorithm described in nvidia
// Image convolution with CUDA for nonseperable kernel
__global__ void convolve2D(CUMATRIX C, CUMATRIX A, CUMATRIX B, int index)
{
  __shared__ float data[3*BLOCK_SIZE][3*BLOCK_SIZE];

  // template size
  int nx = B.dims[1];
  int ny = B.dims[0];

  // feature size
  int fx = A.dims[1];
  int fy = A.dims[0];

  // location in A.data of the current thread
  int x = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  int y = blockIdx.y*BLOCK_SIZE + threadIdx.y - ny/2;

  // load data
  float val;
  if(index == B.dims[2]-1)
    val = 0;
  else
    val = 0;

  int dx = x - BLOCK_SIZE;
  int dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x][threadIdx.y] = val;

  dx = x;
  dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x+BLOCK_SIZE][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x+BLOCK_SIZE][threadIdx.y] = val;

  dx = x + BLOCK_SIZE;
  dy = y;
  if(dx >= 0 && dx < fx && dy >= 0 && dy < fy)
    data[threadIdx.x+2*BLOCK_SIZE][threadIdx.y] = A.data[index*fx*fy+dx*fy+dy];
  else
    data[threadIdx.x+2*BLOCK_SIZE][threadIdx.y] = val;
  __syncthreads();

  if(x < fx && y < fy && threadIdx.y >= ny/2 && threadIdx.y < ny/2 + BLOCK_SIZE)
  {
    // location in shared memory
    int xx = threadIdx.x + BLOCK_SIZE - nx/2;
    int yy = threadIdx.y - ny/2;
    float sum = 0;
    for(int i = 0; i < nx; i++)
    {
      for(int j = 0; j < ny; j++)
        sum += hog_template[i*ny+j] * data[xx+i][yy+j];
    }
    C.data[x*fy+y] += sum;
  }
}

/*
int main(int argc, char** argv)
{
  FILE *fp;
  CUMATRIX A, A_device;
  CUMATRIX B, B_device;
  CUMATRIX C, C_device;
  hipError_t error;

  // load hog features
  fp = fopen(argv[1], "r");
  if(fp == NULL)
  {
    printf("can not open file %s\n", argv[1]);
    return 1;
  }
  A = read_cumatrix(fp);
  fclose(fp);
  A_device = alloc_device_cumatrix(A);

  // generate a random hog template
  B.dims_num = 3;
  B.dims = (int*)malloc(sizeof(int)*3);
  B.dims[0] = 16;
  B.dims[1] = 17;
  B.dims[2] = 32;
  B.length = 16*17*32;
  B.data = (float*)malloc(sizeof(float)*B.length);
  for(int i = 0; i < B.length; i++)
    B.data[i] = 1;
  B_device = alloc_device_cumatrix(B);

  // allocate hog response matrix
  C.dims_num = 2;
  C.dims = (int*)malloc(sizeof(int)*2);
  C.dims[0] = A.dims[0];
  C.dims[1] = A.dims[1];
  C.length = C.dims[0]*C.dims[1];
  C.data = (float*)malloc(sizeof(float)*C.length);
  C_device = alloc_device_cumatrix(C);

  error = hipMemset(C_device.data, 0, sizeof(float)*C_device.length);
  if (error != hipSuccess)
  {
    printf("hipMemset C_device returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

  // setup execution parameters
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE+2*(B.dims[0]/2));
  dim3 grid((C.dims[1]+BLOCK_SIZE-1) / BLOCK_SIZE, (C.dims[0]+BLOCK_SIZE-1) / BLOCK_SIZE);

  for(int i = 0; i < B.dims[2]; i++)
  {
    // copy to constant memory
    error = hipMemcpyToSymbol(HIP_SYMBOL(hog_template), B.data+i*B.dims[0]*B.dims[1], sizeof(float)*B.dims[0]*B.dims[1]);
    if (error != hipSuccess)
    {
      printf("hipMemcpyToSymbol returned error code %d, line(%d)\n", error, __LINE__);
      exit(EXIT_FAILURE);
    }

    convolve2D<<< grid, threads >>>(C_device, A_device, B_device, i);
    hipDeviceSynchronize();
  }

  // copy result from device to host
  error = hipMemcpy(C.data, C_device.data, sizeof(float)*C.length, hipMemcpyDeviceToHost);
  if (error != hipSuccess)
  {
    printf("hipMemcpy C returned error code %d, line(%d)\n", error, __LINE__);
    exit(EXIT_FAILURE);
  }

  fp = fopen(argv[2], "w");
  write_cumatrix(&C, fp);
  fclose(fp);

  free_device_cumatrix(&A_device);
  free_device_cumatrix(&B_device);
  free_device_cumatrix(&C_device);
  free_cumatrix(&A);
  free_cumatrix(&B);
  free_cumatrix(&C);
  return 0;
}
*/
