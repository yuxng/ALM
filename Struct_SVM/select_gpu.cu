/*
  Select a device according to rank
  author: Yu Xiang
  Date: 05/12/2011
*/

extern "C"
{
#include "select_gpu.h"
}
#include <stdio.h>

void select_gpu(int rank)
{
  /* get device count */
  int deviceCount = 0;
  if(hipGetDeviceCount(&deviceCount) != hipSuccess)
  {
    printf("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
    exit(1);
  }
  /*
  deviceCount--;
  int deviceID = rank % deviceCount + 1;
  */
  if(rank == 0)
    printf("%d CUDA enabled devices available.\n", deviceCount);

  int deviceID = rank % deviceCount;
  if(hipSetDevice(deviceID) != hipSuccess)
  {
    printf("hipSetDevice FAILED\n");
    exit(1);
  }
  printf("Process %d is running on GPU %d.\n", rank, deviceID);
}
